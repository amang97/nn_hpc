#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>

#include "matrix.cuh"
#include "nn_layer.cuh"
#include "cuda_utils.cuh"
#include "../nn_param.cuh"

data_t rand_weight() {
    return ((data_t)rand())/((data_t)RAND_MAX);
}

void weight_init(Matrix * W) {
    int row, col;
    for (row = 1; row <= W->rows; row++) {
        for (col = 1; col <= W->cols; col++) {
            ELEMENT(W, row, col) = rand_weight();
        }
    }
    // copy host W to device
    copy_matrix_H2D(W);
}

nnlayer * nnl_init(int l, int Wx, int Wy, char f) {
    nnlayer * nnl = (nnlayer*)malloc(sizeof(nnlayer));
    if (!nnl) { printf("Unabble to initialize nn layer\n"); return NULL; }
    nnl->l = l;
    nnl->A = matrix_init(BATCH_SIZE, Wx);
    nnl->W = matrix_init(Wx, Wy);
    nnl->b = matrix_init(Wy, 1);
    nnl->Z = matrix_init(BATCH_SIZE, Wy);
    nnl->dA = matrix_init(BATCH_SIZE, Wx);
    nnl->dZ = matrix_init(BATCH_SIZE, Wy);
    matrix_allocate(nnl->A);
    matrix_allocate(nnl->W); weight_init(nnl->W);   // initialize random weights
    matrix_allocate(nnl->b);
    matrix_allocate(nnl->Z);
    matrix_allocate(nnl->dA);
    matrix_allocate(nnl->dZ);
    nnl->f = f;
    return nnl;
}

int nnl_free(nnlayer * nnl) {
    if (!nnl) { printf("Unabble to initialize nn layer\n"); return -1; }
    int freea, freew, freeb, freez, freeda, freedz;
    freea = freew = freeb = freez = freeda = freedz = -1;
    if (nnl->A) freea = matrix_free(nnl->A);
    if (nnl->W) freew = matrix_free(nnl->W);
    if (nnl->b) freeb = matrix_free(nnl->b);
    if (nnl->Z) freez = matrix_free(nnl->Z);
    if (nnl->dA) freeda = matrix_free(nnl->dA);
    if (nnl->dZ) freedz = matrix_free(nnl->dZ);
    // printf("A: %d, W: %d, b: %d, Z: %d, dA: %d, dZ: %d\n",
    //        freea, freew, freeb, freez, freeda, freedz);
    if (freea || freew || freeb || freez || freeda || freedz) return -1;
    free(nnl);
    return 0;
}

__global__
void FFNNFP_global(data_t *Z, data_t *W, data_t *A, data_t *b, int Wx, int Wy,
    int Ax, int Ay) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockIdx.x + threadIdx.x;
    int Zx = Ax;
    int Zy = Wy;
    data_t val = (data_t)0;

    int k;
    if (row < Zy && col < Zx) {
        for (k = 0; k < Wx; k++) {
            val += W[row*Wx+k] * A[k*Ax+col];
        }
        Z[row*Zx+col] = val + b[row];
    }
}

/* Forward pass call from host */
Matrix * nnl_forward_pass_global(nnlayer * nnl, Matrix *A) {
    assert(nnl->W->rows == A->cols); nnl->A = A;

    // call forward pass kernel
    dim3 block_W(BLOCK_SIZE_W, BLOCK_SIZE_W);
    dim3 grid_W((nnl->Z->rows+block_W.x-1)/block_W.x,
                (nnl->Z->cols+block_W.y-1)/block_W.y);
    FFNNFP_global<<<grid_W, block_W>>>(nnl->Z->data_d,
                                        nnl->W->data_d,
                                        nnl->A->data_d,
                                        nnl->b->data_d,
                                        nnl->W->rows, nnl->W->cols,
                                        nnl->A->rows, nnl->A->cols);
    return nnl->Z;
}


/* Activations */
/******************************************************************************/
__device__
data_t relu(data_t x, data_t y) {
    return (x > y) ? x : y;
}

__device__
data_t sigmoid(data_t x) {
    return ((data_t)1) / ((data_t)1 + (data_t)exp(-x));
}

__global__
void relu_forward_global(data_t *A, data_t *Z, int Zx, int Zy) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < Zx * Zy) {
        A[index] = relu(Z[index],(data_t)0);
    }
}

__global__
void sigmoid_forward_global(data_t *A, data_t *Z, int Zx, int Zy) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < Zx * Zy) {
        A[index] = sigmoid(Z[index]);
    }
}

/* Host calls to GPU for RELU for forward pass*/
void relu_forward_pass_global(Matrix * A, Matrix * Z) {
    int  Zx = Z->rows, Zy = Z->cols;
    // call relu activation forward pass
    dim3 block(BLOCK_SIZE_b);
    dim3 num_blocks((Zy*Zx+block.x-1)/block.x);
    relu_forward_global<<<num_blocks,block>>>(A->data_d,
                                            Z->data_d,
                                            Zx, Zy);
    // return A;
}

/* Host calls to GPU for Sigmoid for Forward pass */
void sigmoid_forward_pass_global(Matrix * A, Matrix * Z) {
    int Zx = Z->rows; int Zy = Z->cols;

    // call sigmoid activation forward pass
    dim3 block(BLOCK_SIZE_b);
    dim3 num_blocks((Zy*Zx+block.x-1)/block.x);
    sigmoid_forward_global<<<num_blocks,block>>>(A->data_d,
                                                Z->data_d,
                                                Zx, Zy);
    // return A;
}

// /* Testing network and layer initializations */
// printf("On host\n");
// printf("Stats on host for layer %d, Activation: %c\n", nn->layer[0]->l, nn->layer[0]->f);
// print_matrix(nn->layer[0]->A);
// printf("\n");
// print_matrix(nn->layer[0]->W);
// printf("\n");
// print_matrix(nn->layer[0]->b);
// printf("\n");
// print_matrix(nn->layer[0]->Z);
// printf("\n");
// print_matrix(nn->layer[0]->dA);
// printf("\n");
// print_matrix(nn->layer[0]->dZ);
// printf("\n\n");
// printf("Stats on host for layer %d, Activation: %c\n", nn->layer[1]->l, nn->layer[1]->f);
// print_matrix(nn->layer[1]->A);
// printf("\n");
// print_matrix(nn->layer[1]->W);
// printf("\n");
// print_matrix(nn->layer[1]->b);
// printf("\n");
// print_matrix(nn->layer[1]->Z);
// printf("\n");
// print_matrix(nn->layer[1]->dA);
// printf("\n");
// print_matrix(nn->layer[1]->dZ);
// printf("\n\n");

// printf("On Device\n");
// printf("Stats on device for layer %d, Activation: %c\n", nn->layer[0]->l, nn->layer[0]->f);
// print_matrix_d(nn->layer[0]->A);
// printf("\n");
// print_matrix_d(nn->layer[0]->W);
// printf("\n");
// print_matrix_d(nn->layer[0]->b);
// printf("\n");
// print_matrix_d(nn->layer[0]->Z);
// printf("\n");
// print_matrix_d(nn->layer[0]->dA);
// printf("\n");
// print_matrix_d(nn->layer[0]->dZ);
// printf("\n\n");
// printf("Stats on device for layer %d, Activation: %c\n", nn->layer[1]->l, nn->layer[1]->f);
// print_matrix_d(nn->layer[1]->A);
// printf("\n");
// print_matrix_d(nn->layer[1]->W);
// printf("\n");
// print_matrix_d(nn->layer[1]->b);
// printf("\n");
// print_matrix_d(nn->layer[1]->Z);
// printf("\n");
// print_matrix_d(nn->layer[1]->dA);
// printf("\n");
// print_matrix_d(nn->layer[1]->dZ);
// printf("\n\n");