#include "hip/hip_runtime.h"
/* Copyright 2019, Aman Gupta, ENG EC 527, Prof. Martin Herbordt              */
/******************************************************************************/
/* Neural Network Activations library for GPU in C                            */
/******************************************************************************/
/* Libraries */
#include <math.h>
#include "matrix.cuh"
/******************************************************************************/
/* Implementations */
/******************************************************************************/

/* Activations */
/******************************************************************************/
__device__
data_t relu(data_t x, data_t y) {
    return (x <= y) ? x : y;
}

__device__
data_t sigmoid(data_t x) {
    return ((data_t)1) / ((data_t)1 + (data_t)exp(-x));
}

/* RELU Activation Forward pass*/
/******************************************************************************/
__global__
void RELU_forward_global(data_t *A, data_t *Z, int Zx, int Zy) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < Zx * Zy) {
        A[index] = relu(Z[index],(data_t)0);
    }
}

__global__
void RELU_forward_shared(data_t *A, data_t *Z, int Zx, int Zy) {

}

__global__
void RELU_forward_unified(data_t *A, data_t *Z, int Zx, int Zy) {

}

/* Sigmoid Activation Forward Pass*/
/******************************************************************************/
__global__
void Sigmoid_Forward_global(data_t *A, data_t *Z, int Zx, int Zy) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < Zx * Zy) {
        A[index] = sigmoid(Z[index]);
    }
}

__global__
void Sigmoid_Forward_shared(data_t *A, data_t *Z, int Zx, int Zy) {

}

__global__
void Sigmoid_Forward_unified(data_t *A, data_t *Z, int Zx, int Zy) {

}

/* RELU Activation Backward Pass*/
/******************************************************************************/
__global__
void RELU_backward_global(data_t *dZ, data_t *dA, data_t *Z, int Zx, int Zy) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < Zx * zy) {
        dZ[index] = (Z[index] > 0) ? dA[index] : 0;
    }
}

__global__
void RELU_backward_shared(data_t *dZ, data_t *dA, data_t *Z, int Zx, int Zy);

__global__
void RELU_backward_unified(data_t *dZ, data_t *dA, data_t *Z, int Zx, int Zy);

/* Sigmoid Activation Backward Pass*/
/******************************************************************************/
__global__
void Sigmoid_backward_global(data_t *dZ, data_t *dA, data_t *Z, int Zx, int Zy) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < Zx*Zy) {
        dZ[index] = dA[index]*sigmoid(Z[index])*((data_t)1 - sigmoid(Z[index]));
    }
}

__global__
void Sigmoid_backward_shared(data_t *dA, data_t *dZ, data_t *Z, int Zx, int Zy);

__global__
void Sigmoid_backward_unified(data_t *dA, data_t *dZ, data_t *Z, int Zx, int Zy);
