#include "hip/hip_runtime.h"
/* Copyright 2019, Aman Gupta, ENG EC 527, Prof. Martin Herbordt              */
/******************************************************************************/
/* Neural Network Activations library for GPU in C                            */
/******************************************************************************/
/* Libraries */
#include <math.h>
#include "matrix.cuh"
/******************************************************************************/
/* Parameters */
/******************************************************************************/
#define BLOCK_SIZE_b    256
/******************************************************************************/
/* Implementations */
/******************************************************************************/

/* Activations */
/******************************************************************************/
__device__
data_t relu(data_t x, data_t y) {
    return (x <= y) ? x : y;
}

__device__
data_t sigmoid(data_t x) {
    return ((data_t)1) / ((data_t)1 + (data_t)exp(-x));
}

/* RELU Activation Forward pass */
/******************************************************************************/
__global__
void RELU_forward_global(data_t *A, data_t *Z, int Zx, int Zy) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < Zx * Zy) {
        A[index] = relu(Z[index],(data_t)0);
    }
}

__global__
void RELU_forward_shared(data_t *A, data_t *Z, int Zx, int Zy) {

}

__global__
void RELU_forward_unified(data_t *A, data_t *Z, int Zx, int Zy) {

}

/* Sigmoid Activation Forward Pass*/
/******************************************************************************/
__global__
void Sigmoid_Forward_global(data_t *A, data_t *Z, int Zx, int Zy) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < Zx * Zy) {
        A[index] = sigmoid(Z[index]);
    }
}

__global__
void Sigmoid_Forward_shared(data_t *A, data_t *Z, int Zx, int Zy) {

}

__global__
void Sigmoid_Forward_unified(data_t *A, data_t *Z, int Zx, int Zy) {

}

/* RELU Activation Backward Pass*/
/******************************************************************************/
__global__
void RELU_backward_global(data_t *dZ, data_t *dA, data_t *Z, int Zx, int Zy) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < Zx * Zy) {
        dZ[index] = (Z[index] > 0) ? dA[index] : 0;
    }
}

__global__
void RELU_backward_shared(data_t *dZ, data_t *dA, data_t *Z, int Zx, int Zy);

__global__
void RELU_backward_unified(data_t *dZ, data_t *dA, data_t *Z, int Zx, int Zy);

/* Sigmoid Activation Backward Pass*/
/******************************************************************************/
__global__
void Sigmoid_backward_global(data_t *dZ, data_t *dA, data_t *Z, int Zx, int Zy) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < Zx*Zy) {
        dZ[index] = dA[index]*sigmoid(Z[index])*((data_t)1 - sigmoid(Z[index]));
    }
}

__global__
void Sigmoid_backward_shared(data_t *dA, data_t *dZ, data_t *Z, int Zx, int Zy);

__global__
void Sigmoid_backward_unified(data_t *dA, data_t *dZ, data_t *Z, int Zx, int Zy);

/* Host calls to GPU for RELU for forward pass*/
void RELU_forward(layer& l) {
    // assumes Z has been allocated and computed
    int Zx = l.Z->rows; int Zy = l.Z->cols;
    dim3 block(BLOCK_SIZE_b);
    dim3 num_blocks((Zy*Zx+block.x-1)/block.x);
    RELU_forward_global<<<num_blocks,block>>>(l.A->data_d, l.Z->data_d, Zx, Zy);
    // copy results of A from device to host 
    CUDA_SAFE_CALL(hipMemcpy(l.A.data_h, l.A.data_d, Ax*Ay*sizeof(data_t),
    hipMemcpyDeviceToHost));
}

/* Host calls to GPU for RELU for backProp */
void RELU_back_propagation(layer& l, data_t lr) {
    int Zx = l.Z->rows; int Zy = l.Z->cols;
    dim3 block(BLOCK_SIZE_b);
    dim3 num_blocks((Zy*Zx+block.x-1)/block.x);
    RELU_backward_global<<<num_blocks,block>>>(l.dZ->data_d,
                                                l.dA->data_d,
                                                l.Z->data_d,
                                                Zx, Zy);
    // copy results of A from device to host 
    CUDA_SAFE_CALL(hipMemcpy(l.dZ.data_h, l.dZ.data_d, Zx*Zy*sizeof(data_t),
    hipMemcpyDeviceToHost));
}

/* Host calls to GPU for Sigmoid for Forward pass */
void Sigmoid_forward(layer& l) {
    int Zx = l.Z->rows; int Zy = l.Z->cols;
    dim3 block(BLOCK_SIZE_b);
    dim3 num_blocks((Zy*Zx+block.x-1)/block.x);
    Sigmoid_Forward_global<<<num_blocks,block>>>(l.A->data_d, l.Z->data_d, Zx, Zy);    
    // copy results of A from device to host 
    CUDA_SAFE_CALL(hipMemcpy(l.A.data_h, l.A.data_d, Ax*Ay*sizeof(data_t),
    hipMemcpyDeviceToHost));
}

/* Host calls to GPU for Sigmoid for backprop*/
void Sigmoid_back_propagation(layer& l, data_t lr) {
    int Zx = l.Z->rows; int Zy = l.Z->cols;
    dim3 block(BLOCK_SIZE_b);
    dim3 num_blocks((Zy*Zx+block.x-1)/block.x);
    Sigmoid_backward_global<<<num_blocks,block>>>(l.dZ->data_d,
                                                l.dA->data_d,
                                                l.Z->data_d,
                                                Zx, Zy);
    // copy results of A from device to host 
    CUDA_SAFE_CALL(hipMemcpy(l.dZ.data_h, l.dZ.data_d, Zx*Zy*sizeof(data_t),
    hipMemcpyDeviceToHost));
    return (matrix *)l.dZ;
}
