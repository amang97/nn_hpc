#include "hip/hip_runtime.h"
/* Copyright 2019, Aman Gupta, ENG EC 527, Prof. Martin Herbordt              */
/******************************************************************************/
/* Feed Forward Neural Network library for GPU in C                           */
/******************************************************************************/
/* Libraries */
/******************************************************************************/
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "matrix.cuh"
/******************************************************************************/
/* Implementations */
/******************************************************************************/

/* Binary cross Entrpy Loss */
/******************************************************************************/
/* Loss: Binary Cross Entropy (BCE) */
/* Input: prediction array, input data point, its length (number of features)
   Output: Cost = x*logy + (1-x)*log(1-y)
*/
__global__
void BCELoss(data_t *cost, data_t *prediction, data_t *x, int Xdim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < Xdim) {
        // calculate partial cost (pc)
        data_t pc = (x[i]*((data_t)log(prediction[i]))) + 
                    (((data_t)1 - x[i])*((data_t)log((data_t)1-prediction[i])));
        atomicAdd(cost,-pc/Xdim);
    }
}

/* Loss Gradient */
/* Input: prediction, x, and Xdim (number of features)
   Output: gradient stored in dY
*/
__global__
void dBCELoss(data_t *dY, data_t *prediction, data_t *x, int Xdim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < Xdim) {
        dY[i] = (data_t)(-1)*
        (x[i]/prediction[i] - (((data_t)1-x[i])/((data_t)1-prediction[i])));
    }
}