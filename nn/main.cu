#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "matrix.cuh"
#include "cuda_utils.cuh"
#include "nn_param.cuh"

/* Main */
int main() {
    srand(time(NULL));

    // // load training and testing data
    // char * train = (char *)"./mnist/mnist_train.csv";
    // char * test = (char *)"./mnist/mnist_test.csv";

    // printf("\nTraining and Test data Loaded\n");

    // Set GPU Device
    CUDA_SAFE_CALL(hipSetDevice(0));
    
    // initialize matrices
    Matrix *W = matrix_init(100,100);
    matrix_allocate(W,100,100);
    printf("Weights initializedon device and host")
    print_matrix(W);
    int w = matrix_free(W);
    if (!w) printf("Weights freed")
    // // Create a Feed Forward Neural Net (array of layers) and other parameters
    // layer l[NUM_LAYERS];
    // data_t lr = LEARNING_RATE;

    // // initialize layers
    // layer_init(l[0], BATCH_SIZE, NUM_FEATURES, NUM_FEATURES, 60, SEED);
    // layer_init(l[1], NUM_FEATURES, 60, 60, NUM_OUTPUTS, SEED);
    // layer_init(l[2], 60, NUM_OUTPUTS, NUM_OUTPUTS, 1, SEED);
    // printf("\nNeural Net layers initialized\n");
    
    /* Network Training on GPU */

    // // delete layers
    // int l1 = delete_layer(l[0]);
    // int l2 = delete_layer(l[1]);
    // int l3 = delete_layer(l[2]);
    // if (!(l1) || !(l2) || !(l3)) printf("Neural Network layers destroyed\n");
    // else printf("OOpS!, Neural Net destruction went wrong.\n");
    
    return 0;
}
